#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/sort.h>

#define BLOCK_SIZE 32

__device__ unsigned char median(unsigned char* neighborhood, int size)
{
    thrust::sort(thrust::device, neighborhood, neighborhood + size);
    return neighborhood[size / 2];
}

__global__ void median_filter_kernel(unsigned char* output, int width, int height, int kernel_size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= kernel_size / 2 && row < height - kernel_size / 2 && col >= kernel_size / 2 && col < width - kernel_size / 2)
    {
        // Extract the neighborhood
        unsigned char neighborhood[kernel_size * kernel_size];
        for (int i = 0; i < kernel_size; i++)
        {
            for (int j = 0; j < kernel_size; j++)
            {
                neighborhood[i * kernel_size + j] = tex2D<unsigned char>(input_texture, col - kernel_size / 2 + j, row - kernel_size / 2 + i);
            }
        }

        // Compute the median of the neighborhood
        output[row * width + col] = median(neighborhood, kernel_size * kernel_size);
    }
}

void median_filter(const unsigned char* input, unsigned char* output, int width, int height, int kernel_size)
{
    int padding_size = kernel_size / 2;
    int output_width = width - kernel_size + 1;
    int output_height = height - kernel_size + 1;
    size_t input_size = width * height * sizeof(unsigned char);
    size_t output_size = output_width * output_height * sizeof(unsigned char);

    // Allocate memory on the device
    unsigned char* d_output;
    hipMalloc(&d_output, output_size);

    // Copy the input to a texture on the device
    hipArray* d_input_array;
    hipMallocArray(&d_input_array, &input_texture.channelDesc, width, height);
    hipMemcpyToArray(d_input_array, 0, 0, input, input_size, hipMemcpyHostToDevice);
    hipBindTextureToArray(input_texture, d_input_array);

    // Define the grid and block sizes
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((output_width + BLOCK_SIZE - 1) / BLOCK_SIZE, (output_height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Call the kernel
    median_filter_kernel<<<grid_size, block_size>>>(d_output, output_width, output_height, kernel_size);

    // Copy the output to the host
    hipMemcpy(output, d_output, output_size, hipMemcpyDeviceToHost);

    // Free the memory on the device
    hipUnbindTexture(input_texture);
    hipFreeArray(d_input_array);
    hipFree(d_output);
}

int main()
{
    // Load the image
    std::ifstream file("input.jpg", std::ios::binary);
    if (!file)
    {
        std::cerr << "Error: Could not open image file." << std::endl;
        return 1;
    }

    int width = 512;
    int height = 512;
    unsigned char input[width * height];
    file.read(reinterpret_cast<char*>(input), width * height);
    file.close();

    // Apply the median filter with a kernel size of 5
    int kernel_size = 5;
    unsigned char output[(width - kernel_size + 1) * (height - kernel_size + 1)];
    median_filter(input, output, width, height, kernel_size);

    // Write the output image
    std::ofstream output_file("output.jpg", std::ios::binary);
    output_file.write(reinterpret_cast<const char*>(output), (width - kernel_size + 1) * (height - kernel_size + 1));
    output_file.close();

    return 0;
}
